#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <unordered_map>
#include <memory>
#include "defs.h"

using namespace std;

#define BLOCK_SIZE 1024

__device__ bool byteCharsMatch(uint8_t byte, char c1, char c2);

__global__ void matchFile(const uint8_t* file_data, size_t file_len, const char* signature, size_t len, int sigIdx, bool* matchArr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	size_t sig_len = len / 2;
	
	if (idx + sig_len > file_len) {
		return;
	}
	bool match = true;
	for (size_t i = 0; i < sig_len; i++) {
		if (!byteCharsMatch(file_data[idx + i], signature[2*i], signature[2*i+1])) {
			match = false;
			break;
		}
	}
	if (match) {
		matchArr[sigIdx] = 1;
	}
}

void runScanner(std::vector<Signature>& signatures, std::vector<InputFile>& inputs)
{
	{
		hipDeviceProp_t prop;
		check_cuda_error(hipGetDeviceProperties(&prop, 0));

		fprintf(stderr, "cuda stats:\n");
		fprintf(stderr, "  # of SMs: %d\n", prop.multiProcessorCount);
		fprintf(stderr, "  global memory: %.2f MB\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		fprintf(stderr, "  shared mem per block: %zu bytes\n", prop.sharedMemPerBlock);
		fprintf(stderr, "  constant memory: %zu bytes\n", prop.totalConstMem);
	}

	/*
		Here, we are creating one stream per file just for demonstration purposes;
		you should change this to fit your own algorithm and/or implementation.
	*/
	std::vector<hipStream_t> streams {};
	streams.resize(inputs.size());

	std::vector<uint8_t*> file_bufs {};
	for(size_t i = 0; i < inputs.size(); i++)
	{
		hipStreamCreate(&streams[i]);

		// allocate memory on the device for the file
		uint8_t* ptr = 0;
		check_cuda_error(hipMalloc(&ptr, inputs[i].size));
		file_bufs.push_back(ptr);
	}

	// allocate memory for the signatures
	std::vector<char*> sig_bufs {};
	for(size_t i = 0; i < signatures.size(); i++)
	{
		char* ptr = 0;
		check_cuda_error(hipMalloc(&ptr, signatures[i].size));
		hipMemcpy(ptr, signatures[i].data, signatures[i].size, hipMemcpyHostToDevice);
		sig_bufs.push_back(ptr);
	}

	vector<bool*> results;
	results.resize(inputs.size());

	for(size_t file_idx = 0; file_idx < inputs.size(); file_idx++)
	{
		// asynchronously copy the file contents from host memory
		// (the `inputs`) to device memory (file_bufs, which we allocated above)
		hipMemcpyAsync(file_bufs[file_idx], inputs[file_idx].data, inputs[file_idx].size,
			hipMemcpyHostToDevice, streams[file_idx]);    // pass in the stream here to do this async

		bool* matchArr;
		check_cuda_error(hipMallocManaged(&matchArr, sizeof(bool)*signatures.size()));

		for(size_t sig_idx = 0; sig_idx < signatures.size(); sig_idx++)
		{
			size_t n_tasks = (inputs[file_idx].size - (signatures[sig_idx].size)/2 + 1);
			size_t grid_size = (n_tasks + BLOCK_SIZE - 1) / BLOCK_SIZE;

			matchFile<<<grid_size, BLOCK_SIZE, /* shared memory per block: */ 0, streams[file_idx]>>>(
				file_bufs[file_idx], inputs[file_idx].size,
				sig_bufs[sig_idx], signatures[sig_idx].size, sig_idx, matchArr);
		}

		// wait for the stream to finish
		hipStreamSynchronize(streams[file_idx]);

		results[file_idx] = matchArr;
	}

	hipDeviceSynchronize();

	for(size_t file_idx = 0; file_idx < inputs.size(); file_idx++)
	{
		for(size_t sig_idx = 0; sig_idx < signatures.size(); sig_idx++)
		{
			if (results[file_idx][sig_idx]) {
				printf("%s: %s\n", inputs[file_idx].name.c_str(), signatures[sig_idx].name.c_str());
			}
		}
	}

	// free the device memory, though this is not strictly necessary
	// (the CUDA driver will clean up when your program exits)
	for(auto buf : file_bufs)
		hipFree(buf);

	for(auto buf : sig_bufs)
		hipFree(buf);

	// clean up streams (again, not strictly necessary)
	for(auto& s : streams)
		hipStreamDestroy(s);
}

__device__ bool byteCharsMatch(uint8_t byte, char c1, char c2)
{
	const uint8_t c1_byte = c1 >= '0' && c1 <= '9' ? c1 - '0' : c1 - 'a' + 10;
	const uint8_t c2_byte = c2 >= '0' && c2 <= '9' ? c2 - '0' : c2 - 'a' + 10;
	if (c1 == '?' && c2 == '?') {
		return true;
	} else if (c1 == '?') {
		return (byte & 0x0f) == c2_byte;
	} else if (c2 == '?') {
		return (byte >> 4) == c1_byte;
	} else {
		return byte == ((c1_byte << 4) | c2_byte);
	}
}
